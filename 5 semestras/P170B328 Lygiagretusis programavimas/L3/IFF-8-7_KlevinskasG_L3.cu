#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>

using namespace std;


#define THREAD_COUNT 11
#define RESULT_STR_SIZE 30
#define INPUT_FILE "IFF-8-7_KlevinskasG_L3_dat.txt"
#define OUTPUT_FILE "IFF-8-7_KlevinskasG_L3_rez.txt"

typedef struct Car_t {
    char model[10];
    int mileage;
    float fuelEcon;
} Car;


__device__ void mileageToStr(char *dest, int src) {
    int i = 0;
    src /= 1000;

    while (src != 0) {
        int digit = src % 10;
        dest[i++] = '0' + digit;
        src /= 10;
    }

    int start = 0;
    int end = i - 1;

    while (start < end) {
        char temp = dest[start];
        dest[start] = dest[end];
        dest[end] = temp;

        start++;
        end--;
    }

    dest[i++] = 'k';
    dest[i] = '\0';
}


__device__ int slen(const char *str) {
    int len = 0;

    while (str[len] != '\0')
        len++;

    return len;
}


__device__ char *getResultString(Car car) {
    char *result = (char *) malloc(RESULT_STR_SIZE);
    char mileage[10];
    char fuelEconRating;

    mileageToStr(mileage, car.mileage);

    if (car.fuelEcon > 10)
        fuelEconRating = 'E';
    else if (car.fuelEcon > 8)
        fuelEconRating = 'D';
    else if (car.fuelEcon > 6)
        fuelEconRating = 'C';
    else if (car.fuelEcon > 4)
        fuelEconRating = 'B';
    else
        fuelEconRating = 'A';

    int i = slen(car.model);

    memcpy(result, car.model, i);
    result[i++] = '-';
    memcpy(result + i, mileage, slen(mileage));
    i += slen(mileage);
    result[i++] = '-';
    result[i++] = fuelEconRating;
    result[i] = '\0';

    return result;
}


__global__ void getResult(Car *cars, int n, char *results, unsigned int *resultSize) {
    for (int i = threadIdx.x; i < n; i += THREAD_COUNT) {
        char *result = getResultString(cars[i]);

        if (result[0] > 'F') {
            unsigned int insertIndex = atomicInc_system(resultSize, n) * RESULT_STR_SIZE;
            memcpy(results + insertIndex, result, RESULT_STR_SIZE);
        }

        free(result);
    }
}


void writeToFile(char *result, unsigned int n) {
    ofstream out;
    out.open(OUTPUT_FILE);

    if (n == 0) {
        out << "Result array is empty" << endl;
        return;
    }

    for (int i = 0; i < n; i++) {
        out << result + (i * RESULT_STR_SIZE) << endl;
    }

    out << "Size: " << n << endl;
    out.close();
}


int main() {
    ifstream in;
    in.open(INPUT_FILE);

    int n;
    in >> n;

    Car* cars = new Car[n];

    for (int i = 0; i < n; i++) {
        Car car;
        in >> car.mileage >> car.fuelEcon >> car.model;
        cars[i] = car;
    }

    in.close();
    
    Car *deviceCars;
    char *deviceResults;
    unsigned int *deviceResultSize;

    hipMalloc(&deviceCars, n * sizeof(Car));
    hipMalloc(&deviceResults, RESULT_STR_SIZE * n * sizeof(char));
    hipMalloc(&deviceResultSize, sizeof(unsigned int));

    hipMemcpy(deviceCars, cars, n * sizeof(Car), hipMemcpyHostToDevice);
    hipMemset(deviceResultSize, 0, sizeof(unsigned int));

    getResult<<<1, THREAD_COUNT>>>(deviceCars, n, deviceResults, deviceResultSize);
    hipDeviceSynchronize();

    unsigned int resultSize;
    hipMemcpy(&resultSize, deviceResultSize, sizeof(unsigned int), hipMemcpyDeviceToHost);

    if (resultSize == 0) {
        cout << "Result array is empty" << endl;
        writeToFile(nullptr, resultSize);
        return 0;
    }

    char* results = (char*) malloc(RESULT_STR_SIZE * resultSize * sizeof(char));

    for (int i = 0; i < resultSize; i++) {
        hipMemcpy(
            results + (i * RESULT_STR_SIZE),
            deviceResults + (i * RESULT_STR_SIZE),
            RESULT_STR_SIZE * sizeof(char),
            hipMemcpyDeviceToHost
        );
    }

    for (int i = 0; i < resultSize; i++) {
        cout << results + (i * RESULT_STR_SIZE) << endl;
    }

    cout << "Size: " << resultSize << endl;

    writeToFile(results, resultSize);

    delete[] cars;
    hipFree(&deviceCars);
    hipFree(&deviceResults);
    hipFree(&deviceResultSize);
    free(results);

    return 0;
}
